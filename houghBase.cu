#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : Juan Angel Carrera, Juan Carlos Bajan, Jose Mariano Reyes
 Version       : 2.0
 Last modified : November 2023
 License       : Released under the GNU GPL 3.0
 Description   : redo de hough base
 To build use  : make
 To run use    : make run_base
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <vector>
#include "pgm.h"
// tiempo de cuda
#include <hip/hip_runtime.h>

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) 
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa threshold, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

// GPU kernel. One thread per image pixel is spawned.
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  // Calcular gloID teniendo en cuenta la geometría del grid
  int blockID = blockIdx.x + blockIdx.y * gridDim.x;
  int gloID = blockID * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

  if (gloID >= w * h) return; // En caso de hilos extras en el bloque

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  // verifica si la intensidad del píxel en la posición actual del hilo (gloID) en la imagen de entrada (pic) es mayor que 0.
  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;
      atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
    }
  }
}


// main que ejecuta todo el programa
int main(int argc, char **argv)
{
  // variable de threshold, de no otorgarse como parametro es 3115
  int i;
  int threshhold = argv[2] ? atoi(argv[2]) : 3115;
  
  // tiempo
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // create a PGMImage object
  PGMImage* inImg = new PGMImage(argv[1]); 

  // color con el que vamos a colorear la imagen base
  inImg->setColor(0,255,0);

  // dimensiones de la imagen
  int *cpuht;
  int w = inImg->getXDim();
  int h = inImg->getYDim();  

  // asignacion de memoria al seno y cosen
  float *d_Cos;
  float *d_Sin;

  hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
  hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);
 
  // CPU calculation
  CPU_HoughTran(inImg->getPixels(), w, h, &cpuht);

  // Pre-compute values to be stored
  float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
  float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
  float rad = 0;

  // calculo de valores de sen y cosen para varios angulos
  for (int i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos(rad);
    pcSin[i] = sin(rad);
    rad += radInc;
  }

  // calculo de valres con la transformada de hough
  float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // Copiar valores de seno y coseno a la memoria constante
   hipMemcpy(d_Cos, pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
   hipMemcpy(d_Sin, pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
   
  // Setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg->getPixels(); // h_in contiene los pixeles de la imagen

  h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

  hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
  hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
  hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

  // Execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  // 1 thread por pixel
  int blockNum = ceil(w * h / 256.0);
  dim3 gridDim(blockNum, 1);
  dim3 blockDim(16, 16); // Puedes ajustar estos valores según tus necesidades

  // lanzar el kernel y medir el tiempo de ejecución
  hipEventRecord(start, 0);

  // llamada al kernel
  GPU_HoughTran<<<gridDim, blockDim>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  // terminar el timepo de ejecucion
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // Get results from device
  hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // Compare CPU and GPU results
  for (int i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
      printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }

  // codigo  que contiene las coordenadas polares de las líneas detectadas en la imagen, basándose en los resultados de la transformada de Hough
  std::vector<std::pair<int, int>> lines;                                         
  for (i = 0; i < degreeBins * rBins; i++){                                       
    if (h_hough[i] > threshhold) {                                                
      int my_r = i / degreeBins;                                                  
      int my_th = i % degreeBins;                                                 
      std::pair<int, int> line = {my_r, my_th};                                   
      lines.push_back(line);                                                      
    }
  }

  // guardar imagen
  inImg->saveImg("Base.jpeg", lines, radInc, rBins);

  // programa terminado
  printf("Done!\n");

  // tiempo de ejecucion
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Tiempo de ejecución del kernel: %f ms\n", elapsedTime);

  // Liberar memoria
  hipFree(d_in);
  hipFree(d_hough);
  hipFree(d_Cos);
  hipFree(d_Sin);

  free(h_hough);
  free(pcCos);
  free(pcSin);
  delete[] cpuht;

  return 0;
}